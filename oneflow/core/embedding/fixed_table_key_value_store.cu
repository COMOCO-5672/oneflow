/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/embedding/fixed_table_key_value_store.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/embedding/fixed_table.h"
#include <omp.h>
#include <robin_hood.h>
#include <fcntl.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <dirent.h>

namespace oneflow {

namespace embedding {

namespace {

template<typename Key>
class KeyValueStoreImpl : public KeyValueStore {
 public:
  OF_DISALLOW_COPY_AND_MOVE(KeyValueStoreImpl);
  explicit KeyValueStoreImpl(const FixedTableKeyValueStoreOptions& options)
      : device_index_(-1), max_query_length_(options.max_query_length) {
    OF_CUDA_CHECK(hipGetDevice(&device_index_));
    key_size_ = options.table_options.key_size;
    value_size_ = options.table_options.value_size;
    table_ = NewFixedTable(options.table_options);
    OF_CUDA_CHECK(NumaAwareCudaMallocHost(
        device_index_, reinterpret_cast<void**>(&host_query_keys_), key_size_ * max_query_length_));
    OF_CUDA_CHECK(NumaAwareCudaMallocHost(device_index_,
                                          reinterpret_cast<void**>(&host_query_values_),
                                          value_size_ * max_query_length_));
    OF_CUDA_CHECK(NumaAwareCudaMallocHost(device_index_, reinterpret_cast<void**>(&host_n_missing_),
                                          sizeof(uint32_t)));
    OF_CUDA_CHECK(NumaAwareCudaMallocHost(device_index_,
                                          reinterpret_cast<void**>(&host_missing_keys_),
                                          key_size_ * max_query_length_));
    OF_CUDA_CHECK(NumaAwareCudaMallocHost(device_index_,
                                          reinterpret_cast<void**>(&host_missing_indices_),
                                          sizeof(uint32_t) * max_query_length_));
  }
  ~KeyValueStoreImpl() {
    CudaCurrentDeviceGuard guard(device_index_);
    OF_CUDA_CHECK(hipHostFree(host_query_keys_));
    OF_CUDA_CHECK(hipHostFree(host_query_values_));
  }

  uint32_t KeySize() const override { return key_size_; }

  uint32_t ValueSize() const override { return value_size_; }

  uint32_t MaxQueryLength() const override { return GetMaxVal<int32_t>(); }

  void Get(ep::Stream* stream, uint32_t num_keys, const void* keys, void* values,
           uint32_t* n_missing, void* missing_keys, uint32_t* missing_indices,
           uint64_t* context) override;
  void Put(ep::Stream* stream, uint32_t num_keys, const void* keys, const void* values,
           uint64_t* context) override;

 private:
  void LoadSnapshot(const std::string& name) override;
  void SaveSnapshot(const std::string& name) override;

  int device_index_;
  uint32_t max_query_length_;
  uint32_t key_size_;
  uint32_t value_size_;
  Key* host_query_keys_{};
  uint8_t* host_query_values_{};
  uint32_t* host_n_missing_{};
  Key* host_missing_keys_{};
  uint32_t* host_missing_indices_{};

  std::mutex mutex_;
  std::unique_ptr<FixedTable> table_;
};

template<typename Key>
void KeyValueStoreImpl<Key>::Get(ep::Stream* stream, uint32_t num_keys, const void* keys,
                                 void* values, uint32_t* n_missing, void* missing_keys,
                                 uint32_t* missing_indices, uint64_t* context) {
  std::lock_guard<std::mutex> lock(mutex_);
  auto hip_stream = stream->As<ep::CudaStream>();
  CHECK_LE(num_keys, max_query_length_);
  if (num_keys == 0) {
    OF_CUDA_CHECK(hipMemsetAsync(n_missing, 0, sizeof(uint32_t),
                                  stream->As<ep::CudaStream>()->hip_stream()));
    return;
  }
  OF_CUDA_CHECK(hipMemcpyAsync(host_query_keys_, keys, key_size_ * num_keys, hipMemcpyDefault,
                                hip_stream->hip_stream()));
  CHECK_JUST(hip_stream->Sync());

  table_->Get(num_keys, host_query_keys_, host_query_values_, host_n_missing_,
              host_missing_indices_);

  OF_CUDA_CHECK(hipMemcpyAsync(values, host_query_values_, num_keys * value_size_,
                                hipMemcpyDefault, hip_stream->hip_stream()));
  OF_CUDA_CHECK(hipMemcpyAsync(n_missing, host_n_missing_, sizeof(uint32_t), hipMemcpyDefault,
                                hip_stream->hip_stream()));
  OF_CUDA_CHECK(hipMemcpyAsync(missing_keys, host_missing_keys_, (*host_n_missing_) * key_size_,
                                hipMemcpyDefault, hip_stream->hip_stream()));
  OF_CUDA_CHECK(hipMemcpyAsync(missing_indices, host_missing_indices_,
                                (*host_n_missing_) * sizeof(uint32_t), hipMemcpyDefault,
                                hip_stream->hip_stream()));
}

template<typename Key>
void KeyValueStoreImpl<Key>::Put(ep::Stream* stream, uint32_t num_keys, const void* keys,
                                 const void* values, uint64_t* context) {
  std::lock_guard<std::mutex> lock(mutex_);
  auto hip_stream = stream->As<ep::CudaStream>();
  CHECK_LE(num_keys, max_query_length_);
  if (num_keys == 0) { return; }
  OF_CUDA_CHECK(hipMemcpyAsync(host_query_keys_, keys, key_size_ * num_keys, hipMemcpyDefault,
                                hip_stream->hip_stream()));
  OF_CUDA_CHECK(hipMemcpyAsync(host_query_values_, values, value_size_ * num_keys,
                                hipMemcpyDefault, hip_stream->hip_stream()));
  CHECK_JUST(hip_stream->Sync());
  table_->Put(num_keys, host_query_keys_, host_query_values_);
}

template<typename Key>
void KeyValueStoreImpl<Key>::LoadSnapshot(const std::string& name) {
  table_->LoadSnapshot(name);
}

template<typename Key>
void KeyValueStoreImpl<Key>::SaveSnapshot(const std::string& name) {
  table_->SaveSnapshot(name);
}

}  // namespace

std::unique_ptr<KeyValueStore> NewFixedTableKeyValueStore(
    const FixedTableKeyValueStoreOptions& options) {
  return std::unique_ptr<KeyValueStore>(new KeyValueStoreImpl<uint64_t>(options));
}

}  // namespace embedding

}  // namespace oneflow
