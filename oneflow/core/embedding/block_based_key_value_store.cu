/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/embedding/block_based_key_value_store.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/embedding/file_handle.h"
#include "oneflow/core/embedding/fixed_table.h"
#include <omp.h>
#include <robin_hood.h>
#include <fcntl.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <dirent.h>

namespace oneflow {

namespace embedding {

namespace {

constexpr uint64_t NUM_BLOCKS_PER_CHUNK = 4 * 1024 * 1024;

template<typename Key>
class KeyValueStoreImpl : public KeyValueStore {
 public:
  OF_DISALLOW_COPY_AND_MOVE(KeyValueStoreImpl);
  explicit KeyValueStoreImpl(const BlockBasedKeyValueStoreOptions& options)
      : device_index_(-1),
        value_length_(options.value_length),
        max_query_length_(options.max_query_length),
        block_size_(options.block_size) {
    OF_CUDA_CHECK(hipGetDevice(&device_index_));
    key_size_ = GetSizeOfDataType(options.key_type);
    value_size_ = GetSizeOfDataType(options.value_type) * value_length_;

    FixedTableOptions table_options{};
    table_options.path = options.path;
    table_options.key_size = key_size_;
    table_options.value_size = value_size_;
    table_options.num_blocks_per_chunk = NUM_BLOCKS_PER_CHUNK;
    table_options.block_size = block_size_;
    table_ = NewFixedTable(table_options);
    CHECK_GE(block_size_, value_size_);
    OF_CUDA_CHECK(NumaAwareCudaMallocHost(
        device_index_, reinterpret_cast<void**>(&host_query_keys_), key_size_ * max_query_length_));
    OF_CUDA_CHECK(NumaAwareCudaMallocHost(device_index_,
                                          reinterpret_cast<void**>(&host_query_values_),
                                          value_size_ * max_query_length_));
    OF_CUDA_CHECK(NumaAwareCudaMallocHost(device_index_, reinterpret_cast<void**>(&host_n_missing_),
                                          sizeof(uint32_t)));
    OF_CUDA_CHECK(NumaAwareCudaMallocHost(device_index_,
                                          reinterpret_cast<void**>(&host_missing_keys_),
                                          key_size_ * max_query_length_));
    OF_CUDA_CHECK(NumaAwareCudaMallocHost(device_index_,
                                          reinterpret_cast<void**>(&host_missing_indices_),
                                          sizeof(uint32_t) * max_query_length_));
  }
  ~KeyValueStoreImpl() {
    CudaCurrentDeviceGuard guard(device_index_);
    OF_CUDA_CHECK(hipHostFree(host_query_keys_));
    OF_CUDA_CHECK(hipHostFree(host_query_values_));
  }

  void Get(ep::Stream* stream, uint32_t num_keys, const void* keys, void* values,
           uint32_t* n_missing, void* missing_keys, uint32_t* missing_indices,
           uint64_t* context) override;
  void Put(ep::Stream* stream, uint32_t num_keys, const void* keys, const void* values,
           uint64_t* context) override;

 private:
  void LoadSnapshot(const std::string& name) override;
  void SaveSnapshot(const std::string& name) override;

  int device_index_;
  uint32_t value_length_;
  uint32_t max_query_length_;
  uint32_t key_size_;
  uint32_t value_size_;
  Key* host_query_keys_{};
  uint8_t* host_query_values_{};
  uint32_t* host_n_missing_{};
  Key* host_missing_keys_{};
  uint32_t* host_missing_indices_{};
  uint64_t block_size_;

  std::mutex mutex_;
  std::unique_ptr<FixedTable> table_;
};

template<typename Key>
void KeyValueStoreImpl<Key>::Get(ep::Stream* stream, uint32_t num_keys, const void* keys,
                                 void* values, uint32_t* n_missing, void* missing_keys,
                                 uint32_t* missing_indices, uint64_t* context) {
  std::lock_guard<std::mutex> lock(mutex_);
  auto hip_stream = stream->As<ep::CudaStream>();
  OF_CUDA_CHECK(hipMemcpyAsync(host_query_values_, values, num_keys * value_size_,
                                hipMemcpyDefault, hip_stream->hip_stream()));
  CHECK_LE(num_keys, max_query_length_);
  if (num_keys == 0) {
    OF_CUDA_CHECK(hipMemsetAsync(n_missing, 0, sizeof(uint32_t),
                                  stream->As<ep::CudaStream>()->hip_stream()));
    return;
  }
  OF_CUDA_CHECK(hipMemcpyAsync(host_query_keys_, keys, key_size_ * num_keys, hipMemcpyDefault,
                                hip_stream->hip_stream()));
  CHECK_JUST(hip_stream->Sync());

  table_->Get(num_keys, host_query_keys_, host_query_values_, host_n_missing_,
              host_missing_indices_);

  OF_CUDA_CHECK(hipMemcpyAsync(values, host_query_values_, num_keys * value_size_,
                                hipMemcpyDefault, hip_stream->hip_stream()));
  OF_CUDA_CHECK(hipMemcpyAsync(n_missing, host_n_missing_, sizeof(uint32_t), hipMemcpyDefault,
                                hip_stream->hip_stream()));
  OF_CUDA_CHECK(hipMemcpyAsync(missing_keys, host_missing_keys_, (*host_n_missing_) * key_size_,
                                hipMemcpyDefault, hip_stream->hip_stream()));
  OF_CUDA_CHECK(hipMemcpyAsync(missing_indices, host_missing_indices_,
                                (*host_n_missing_) * sizeof(uint32_t), hipMemcpyDefault,
                                hip_stream->hip_stream()));
}

template<typename Key>
void KeyValueStoreImpl<Key>::Put(ep::Stream* stream, uint32_t num_keys, const void* keys,
                                 const void* values, uint64_t* context) {
  std::lock_guard<std::mutex> lock(mutex_);
  auto hip_stream = stream->As<ep::CudaStream>();
  CHECK_LE(num_keys, max_query_length_);
  if (num_keys == 0) { return; }
  OF_CUDA_CHECK(hipMemcpyAsync(host_query_keys_, keys, key_size_ * num_keys, hipMemcpyDefault,
                                hip_stream->hip_stream()));
  OF_CUDA_CHECK(hipMemcpyAsync(host_query_values_, values, value_size_ * num_keys,
                                hipMemcpyDefault, hip_stream->hip_stream()));
  CHECK_JUST(hip_stream->Sync());
  table_->Put(num_keys, host_query_keys_, host_query_values_);
}

template<typename Key>
void KeyValueStoreImpl<Key>::LoadSnapshot(const std::string& name) {
  table_->LoadSnapshot(name);
}

template<typename Key>
void KeyValueStoreImpl<Key>::SaveSnapshot(const std::string& name) {
  table_->SaveSnapshot(name);
}

}  // namespace

std::unique_ptr<KeyValueStore> NewBlockBasedKeyValueStore(
    const BlockBasedKeyValueStoreOptions& options) {
  return std::unique_ptr<KeyValueStore>(new KeyValueStoreImpl<uint64_t>(options));
}

}  // namespace embedding

}  // namespace oneflow
