#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include <cstdint>
#include "oneflow/core/common/just.h"
#include "oneflow/core/common/util.h"
#include "oneflow/core/framework/consistency_check.h"
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/kernel/new_kernel_util.h"
#include "oneflow/core/kernel/kernel_util.h"
#include "oneflow/core/ep/cuda/hip_stream.h"
#include "oneflow/core/common/nd_index_offset_helper.h"

namespace oneflow {

namespace {

constexpr size_t NUM_DIM = 8;

template<typename T>
__global__ void AsStrided_kernel(const T* input_buf, T* output_buf, const int64_t* dest_dims, const int32_t* stride,
                const int32_t dest_num_dims, const int32_t storage_offset, const int32_t input_num, const int32_t output_num) {
  NdIndexOffsetHelper<int64_t,NUM_DIM> destIndexOffsetHelper(dest_dims, dest_num_dims);
  CUDA_1D_KERNEL_LOOP_T(int64_t, i, output_num) {
     int64_t dst_index[NUM_DIM];
     destIndexOffsetHelper.OffsetToNdIndex(i, dst_index, dest_num_dims);
     int32_t index_in_input = storage_offset;
     FOR_RANGE(int64_t, j, 0, dest_num_dims){
        index_in_input += dst_index[0];
        //index_in_input+=dst_index[j]*stride[j];
     }
     output_buf[i] = input_buf[index_in_input];
  }
}

template<typename T>
__global__ void AsStridedGrad_kernel(const T* dy_buf, T* dx_buf, const int64_t* dy_dims, const int32_t* stride,
                const int32_t dy_num_dims, const int32_t storage_offset, const int32_t dx_num, const int32_t dy_num) {
    NdIndexOffsetHelper<int64_t,NUM_DIM> destIndexOffsetHelper(dy_dims, dy_num_dims);
    CUDA_1D_KERNEL_LOOP_T(int64_t, i, dy_num) {
        int64_t dy_index[NUM_DIM];
        destIndexOffsetHelper.OffsetToNdIndex(i, dy_index,dy_num_dims);
        int32_t index_in_dx = storage_offset;
        FOR_RANGE(int64_t, j, 0, dy_num_dims){
            index_in_dx+=dy_index[j]*stride[j];
        }
        dx_buf[index_in_dx] += dy_buf[i];
    }
}

template<typename T>
struct AsStridedFunctor final {
void operator()(ep::Stream* stream, const T* input_buf, T* output_buf, const int64_t* dest_dims, const int32_t* stride,
                const int32_t dest_num_dims, const int32_t storage_offset, const int32_t input_num, const int32_t output_num) {
    AsStrided_kernel<T><<<BlocksNum4ThreadsNum(output_num), kCudaThreadsNumPerBlock, 0,
           stream->As<ep::CudaStream>()->hip_stream()>>>(input_buf, output_buf, dest_dims, stride,
           dest_num_dims, storage_offset, input_num, output_num);
}
};

template<typename T>
struct AsStridedGradFunctor final {
 void operator()(ep::Stream* stream, const T* dy_buf, T* dx_buf, const int64_t* dy_dims, const int32_t* stride,
                const int32_t dy_num_dims, const int32_t storage_offset, const int32_t dx_num, const int32_t dy_num) {
    AsStridedGrad_kernel<T><<<BlocksNum4ThreadsNum(dy_num), kCudaThreadsNumPerBlock, 0,
           stream->As<ep::CudaStream>()->hip_stream()>>>(dy_buf, dx_buf, dy_dims, stride,
                dy_num_dims, storage_offset, dx_num, dy_num);
}
};

}

template<typename T>
class GpuAsStridedKernel final : public user_op::OpKernel {
 public:
  GpuAsStridedKernel() = default;
  ~GpuAsStridedKernel() = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* input = ctx->Tensor4ArgNameAndIndex("input", 0);
    user_op::Tensor* output = ctx->Tensor4ArgNameAndIndex("output", 0);
    const auto size = ctx->Attr<std::vector<int32_t>>("size");
    const auto stride = ctx->Attr<std::vector<int32_t>>("stride");
    const int32_t storage_offset = ctx->Attr<int32_t>("storage_offset");
    
    size_t dest_num_dims = output->shape().NumAxes();
    const int64_t *dest_dims = output->shape().ptr();
    const size_t input_num = input->shape().Count(0);
    const size_t output_num = output->shape().Count(0);

    AsStridedFunctor<T>()(ctx->stream(), input->dptr<T>(), output->mut_dptr<T>(), dest_dims, stride.data(), dest_num_dims, storage_offset,
                          input_num, output_num);
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

template<typename T>
class GpuAsStridedGradKernel final : public user_op::OpKernel {
 public:
  GpuAsStridedGradKernel() = default;
  ~GpuAsStridedGradKernel() = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* dy = ctx->Tensor4ArgNameAndIndex("dy", 0);
    user_op::Tensor* dx = ctx->Tensor4ArgNameAndIndex("dx", 0);
    const auto size = ctx->Attr<std::vector<int32_t>>("size");
    const auto stride = ctx->Attr<std::vector<int32_t>>("stride");
    const int32_t storage_offset = ctx->Attr<int32_t>("storage_offset");

    size_t dy_num_dims = dy->shape().NumAxes();
    const int64_t *dy_dims = dy->shape().ptr();
    const size_t dx_num = dx->shape().Count(0);
    const size_t dy_num = dy->shape().Count(0);
    
    Memset<DeviceType::kCPU>(ctx->stream(), dx->mut_dptr(), 0, dx->shape().Count(0) * sizeof(T));
    
    AsStridedGradFunctor<T>()(ctx->stream(), dy->dptr<T>(), dx->mut_dptr<T>(), dy_dims, stride.data(), dy_num_dims, storage_offset,
                          dx_num, dy_num);

  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};



#define REGISTER_GPUASSTRIDED_KERNEL(in_type)                                     \
    REGISTER_USER_KERNEL("as_strided")                         \
      .SetCreateFn<                                            \
          GpuAsStridedKernel<in_type>>()                       \
      .SetIsMatchedHob(                                        \
          (user_op::HobDeviceType() == DeviceType::kCUDA)       \
          && (user_op::HobDataType("input", 0) == GetDataType<in_type>::value));   \
    REGISTER_USER_KERNEL("as_strided_grad")                    \
      .SetCreateFn<                                            \
          GpuAsStridedGradKernel<in_type>>()                   \
      .SetIsMatchedHob(                                        \
          (user_op::HobDeviceType() == DeviceType::kCUDA)       \
          && (user_op::HobDataType("input", 0) == GetDataType<in_type>::value));            

REGISTER_GPUASSTRIDED_KERNEL(float);
REGISTER_GPUASSTRIDED_KERNEL(double);
REGISTER_GPUASSTRIDED_KERNEL(int8_t);
REGISTER_GPUASSTRIDED_KERNEL(int32_t);
REGISTER_GPUASSTRIDED_KERNEL(int64_t);

#undef  REGISTER_GPUASSTRIDED_KERNEL

}  // namespace oneflow